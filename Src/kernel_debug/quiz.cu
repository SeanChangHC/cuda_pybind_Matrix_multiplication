#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper functions for CUDA error checking and initialization

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. A 2D thread block and 2D grid are used. sumArraysOnHost sequentially
 * iterates through vector elements on the host.
 */

void initialData(float *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx,
                     const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];

        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}


void checkCudaErrorsResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %f gpu %f\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

// grid 2D block 2D
__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, int nx,
                                 int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        MatA[idx] += MatB[idx];
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    checkCudaErrors(hipSetDevice(dev));

    // set up data size of matrix
	int nx = 1 << 14;
	int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nxy);
	initialData(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    checkCudaErrors(hipMalloc((void **)&d_MatA, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_MatB, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    checkCudaErrors(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, nx, ny);
    checkCudaErrors(hipDeviceSynchronize());

    // checkCudaErrors kernel error
    checkCudaErrors(hipGetLastError());

    // copy kernel result back to host side
    checkCudaErrors(hipMemcpy(gpuRef, d_MatA, nBytes, hipMemcpyDeviceToHost));

    // checkCudaErrors device results
    checkCudaErrorsResult(hostRef, gpuRef, nxy);

    // free device global memory
    checkCudaErrors(hipFree(d_MatA));
    checkCudaErrors(hipFree(d_MatB));
    checkCudaErrors(hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    checkCudaErrors(hipDeviceReset());

    return (0);
}
